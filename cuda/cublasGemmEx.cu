#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hipblas.h"
#include <iostream>
#include <math.h>
 
using namespace std;
 
#define M 5
#define N 3
#define BYTE 128
 
int main() 
{   
      // 定义状态变量，矩阵乘法接口的返回值
      hipblasStatus_t status;
  
      // 在 CPU内存 中为将要计算的矩阵开辟空间
      float *h_A = (float *)malloc (N * M * sizeof(float));
      char *h_B = (char *)malloc (N * M * sizeof(char));
      
      // 在 CPU内存 中为将要存放运算结果的矩阵开辟空间
      float *h_C = (float *)malloc ( M * M * sizeof(float));
  
      float *f_A = (float *)malloc(N * M * sizeof(float));
      float *f_B = (float *)malloc(N * M * sizeof(float));
      // 为待运算矩阵的元素赋予 0-10 范围内的随机数，实际使用时，A矩阵需要做转置，B矩阵不需要转置，可以手动转置也可以算法内转置
      for (int i = 0; i < N * M; i++) 
      {
          f_A[i] = (float)(rand()%10+1);
          f_B[i] = (float)(rand()%10+1);
      }
 
      //FP32转int8型，有精度损失
      for(int i = 0; i < N * M; i++)
      {
//          h_A[i] = (char)(round(f_A[i] * BYTE)); 
          h_A[i] = f_A[i];
          h_B[i] = (char)(round(f_B[i] * BYTE)); 
      }
      
      // 打印待测试的矩阵
      cout << "矩阵 A :" << endl;
      for (int i = 0; i < N * M; i++){
          cout << (float)h_A[i] << " ";
          if ((i + 1) % N == 0) 
              cout << endl;
      }
      cout << endl;
      cout << "矩阵 B :" << endl;
      for (int i = 0; i < N * M; i++){
         cout << (int)h_B[i] << " ";
          if ((i + 1) % M == 0) 
              cout << endl;
      }
      cout << endl;
      
      /*
      ** GPU 计算矩阵相乘
      */
  
      // 创建并初始化 CUBLAS 库对象
      hipblasHandle_t handle;
      status = hipblasCreate(&handle);
      
      if (status != HIPBLAS_STATUS_SUCCESS)
      {
          if (status == HIPBLAS_STATUS_NOT_INITIALIZED) {
              cout << "CUBLAS 对象实例化出错" << endl;
          }
          getchar ();
          return EXIT_FAILURE;
      }
  
      float *d_A;
      char *d_B;
      float *d_C;
      // 在 显存 中为将要计算的矩阵开辟空间
      hipMalloc (
          (void **)&d_A,    // 指向开辟的空间的指针
          N * M * sizeof(float)    //　需要开辟空间的字节数
      );
      hipMalloc (
          (void **)&d_B,    
          N * M * sizeof(char)    
      );
  
      // 在 显存 中为将要存放运算结果的矩阵开辟空间
      hipMalloc (
          (void **)&d_C,
          M * M * sizeof(float)    
      );
  
      // 将矩阵数据传递进 显存 中已经开辟好了的空间
      hipblasSetVector (
          N * M,    // 要存入显存的元素个数
          sizeof(float),    // 每个元素大小
          h_A,    // 主机端起始地址
          1,    // 连续元素之间的存储间隔
          d_A,    // GPU 端起始地址
          1    // 连续元素之间的存储间隔
      );
     //注意：当矩阵过大时，使用cudaMemcpy是更好地选择：
     //cudaMemcpy(d_A, h_A, sizeof(char)*N*M, cudaMemcpyHostToDevice);
 
     hipblasSetVector (
          N * M, 
          sizeof(char), 
          h_B, 
          1, 
          d_B, 
          1
      );
     //cudaMemcpy(d_B, h_B, sizeof(char)*N*M, cudaMemcpyHostToDevice);
     // 同步函数
//     cudaThreadSynchronize();
     hipDeviceSynchronize(); 
     // 传递进矩阵相乘函数中的参数，具体含义请参考函数手册。
     float a = 1.0; 
     float b = 0;
     // 矩阵相乘。该函数必然将数组解析成列优先数组
//     cublasSgemm (
//         handle,    // blas 库对象 
//         CUBLAS_OP_T,    // 矩阵 A 属性参数
//         CUBLAS_OP_T,    // 矩阵 B 属性参数
//         M,    // A, C 的行数 
//         M,    // B, C 的列数
//         N,    // A 的列数和 B 的行数
//         &a,    // 运算式的 α 值
//         d_A,    // A 在显存中的地址
//         N,    // lda
//         d_B,    // B 在显存中的地址
//         M,    // ldb
//         &b,    // 运算式的 β 值
//         d_C,    // C 在显存中的地址(结果矩阵)
//         M    // ldc
//     );
    hipblasGemmEx(handle,               //句柄
                   HIPBLAS_OP_T,         //矩阵 A 属性参数
                   HIPBLAS_OP_T,         //矩阵 B 属性参数
                   M,                   //A, C 的行数 
                   M,                   //B, C 的列数
                   N,                   //A 的列数和 B 的行数
                   &a,                  //运算式的 α 值
                   d_A,                 //A矩阵
                   HIP_R_32F,           //A矩阵计算模式，FP32型
                   N,                   //A矩阵的列数
                   d_B,                 //B矩阵
                   HIP_R_8I,           //B矩阵计算模式，int8型
                   M,                   //B矩阵的行数
                   &b,                  //乘法因子beta
                   d_C,                 //C结果矩阵
                   HIP_R_32F,          //C矩阵计算模式，FP3232型
                   M,                   //C矩阵的行数
                   HIP_R_32F,          //计算模式，FP32模式
                   CUBLAS_GEMM_ALGO0    //算法参数
                   ); 
     
     // 同步函数
     //cudaDeviceSynchronize();
     hipDeviceSynchronize(); 
     // 从 显存 中取出运算结果至 内存中去
     hipblasGetVector (
         M * M,    //  要取出元素的个数
         sizeof(float),    // 每个元素大小
         d_C,    // GPU 端起始地址
         1,    // 连续元素之间的存储间隔
         h_C,    // 主机端起始地址
         1    // 连续元素之间的存储间隔
     );
 
     //或使用cudaMemcpy(h_C, d_C, sizeof(int)*M*M, cudaMemcpyDeviceToHost);
     // 打印运算结果
     cout << "计算结果的转置 ( (A*B)的转置 )：" << endl;
 
     for (int i = 0;i < M * M; i++)
     {
          cout << h_C[i] << " ";
          if ((i+1)%M == 0) 
            cout << endl;
     }
     
     //注意，这里需要进行归一化操作，乘出来的矩阵需要除以128*128，以还原原来的大小。在此就省略这一步。
     // 清理掉使用过的内存
     free (h_C);
     free (h_B);
     free (h_A);
     free (f_B);
     free (f_A);
 
     try
     {
         hipFree (d_A);
         hipFree (d_B);
         hipFree (d_C);
     }
     catch(...)
     {
          cout << "hipFree Error!" << endl;
          // 释放 CUBLAS 库对象
     }
 
     hipblasDestroy (handle);
     return 0;
}
